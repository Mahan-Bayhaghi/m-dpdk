#include <rte_eal.h>
#include <rte_ethdev.h>
#include <rte_mbuf.h>
#include <rte_mempool.h>
#include <rte_errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TX_RING_SIZE 512
#define NUM_MBUFS (16 * 1024)
#define MBUF_CACHE_SIZE 32
#define BURST_SIZE 256
#define TOTAL_PACKETS (10612 - 1)

static const struct rte_eth_conf port_conf_default = {
    .rxmode = {.max_lro_pkt_size = RTE_ETHER_MAX_LEN}
};

// Define ping-pong buffer structure
#define PING_PONG_SIZE 256
struct ping_pong_buffer {
    struct rte_mbuf *pkts[PING_PONG_SIZE];
    uint16_t count;
};

// Function to perform Host to Device (HD) transfer
void host_to_device(struct ping_pong_buffer *buffer, char *d_buffer, size_t buffer_size) {
    // Allocate memory for packet data
    size_t total_data_size = 0;
    for (int i = 0; i < buffer->count; i++) {
        total_data_size += rte_pktmbuf_pkt_len(buffer->pkts[i]);
    }
    if (total_data_size > buffer_size) {
        printf("Error: Not enough device memory\n");
        return;
    }

    // Copy packet data to a host buffer
    char *h_buffer = (char *)malloc(total_data_size);
    char *cur_ptr = h_buffer;
    for (int i = 0; i < buffer->count; i++) {
        size_t pkt_len = rte_pktmbuf_pkt_len(buffer->pkts[i]);
        rte_memcpy(cur_ptr, rte_pktmbuf_mtod(buffer->pkts[i], const void*), pkt_len);
        cur_ptr += pkt_len;
    }

    // Copy data from host to device (H2D)
    hipMemcpy(d_buffer, h_buffer, total_data_size, hipMemcpyHostToDevice);

    // Free the host buffer
    free(h_buffer);

    // Process or send packets on the device (can be replaced with CUDA kernel launch)
    printf("Transferred %lu bytes from Host to Device\n", total_data_size);

    // Reset the ping-pong buffer after transfer
    for (int i = 0; i < buffer->count; i++) {
        rte_pktmbuf_free(buffer->pkts[i]);
    }
    buffer->count = 0;
}

// Deep copy mbuf into the ping-pong buffer
struct rte_mbuf *deep_copy_mbuf(struct rte_mbuf *src_mbuf, struct rte_mempool *mbuf_pool) {
    struct rte_mbuf *dst_mbuf = rte_pktmbuf_alloc(mbuf_pool); // Allocate a new mbuf
    if (dst_mbuf == NULL) {
        printf("Mbuf allocation failed\n");
        return NULL;
    }
    // Copy packet data and metadata (header + payload)
    if (rte_pktmbuf_pkt_len(src_mbuf) > rte_pktmbuf_tailroom(dst_mbuf)) {
        printf("Not enough space in the destination mbuf\n");
        rte_pktmbuf_free(dst_mbuf);
        return NULL;
    }
    // Copy the packet payload
    rte_memcpy(rte_pktmbuf_mtod(dst_mbuf, void*),
               rte_pktmbuf_mtod(src_mbuf, const void*),
               rte_pktmbuf_pkt_len(src_mbuf));
    // Set the length and metadata
    rte_pktmbuf_pkt_len(dst_mbuf) = rte_pktmbuf_pkt_len(src_mbuf);
    rte_pktmbuf_data_len(dst_mbuf) = rte_pktmbuf_data_len(src_mbuf);
    return dst_mbuf;
}

int main(int argc, char *argv[]) {
    int RX_RING_SIZE;
    int sleep_ms;
    FILE *file;
    file = fopen("config", "r");
    if (file == NULL) {
        perror("Failed to open config file");
        return 1;
    }
    if (fscanf(file, "%d", &RX_RING_SIZE) != 1 || fscanf(file, "%d", &sleep_ms) != 1) {
        fprintf(stderr, "Failed to read from config file\n");
        fclose(file);
        return 1;
    }
    fclose(file);

    struct rte_mempool *mbuf_pool;
    struct rte_eth_dev_info dev_info;
    struct rte_eth_stats eth_stats;
    uint16_t port_id = 0;

    if (rte_eal_init(argc, argv) < 0)
        rte_exit(EXIT_FAILURE, "Error with EAL initialization\n");

    mbuf_pool = rte_pktmbuf_pool_create("MBUF_POOL", NUM_MBUFS,
                                        MBUF_CACHE_SIZE, 0, RTE_MBUF_DEFAULT_BUF_SIZE,
                                        rte_socket_id());
    if (mbuf_pool == NULL)
        rte_exit(EXIT_FAILURE, "Cannot create mbuf pool\n");

    if (rte_eth_dev_info_get(port_id, &dev_info) != 0)
        rte_exit(EXIT_FAILURE, "Error getting device info\n");

    if (rte_eth_dev_configure(port_id, 1, 1, &port_conf_default) != 0)
        rte_exit(EXIT_FAILURE, "Cannot configure device\n");

    if (rte_eth_rx_queue_setup(port_id, 0, RX_RING_SIZE, rte_eth_dev_socket_id(port_id), NULL, mbuf_pool) != 0)
        rte_exit(EXIT_FAILURE, "Cannot setup RX queue\n");

    if (rte_eth_tx_queue_setup(port_id, 0, TX_RING_SIZE, rte_eth_dev_socket_id(port_id), NULL) != 0)
        rte_exit(EXIT_FAILURE, "Cannot setup TX queue\n");

    if (rte_eth_dev_start(port_id) != 0)
        rte_exit(EXIT_FAILURE, "Cannot start device\n");

    if (rte_eth_promiscuous_enable(port_id) != 0)
        rte_exit(EXIT_FAILURE, "Cannot enable promiscuous mode\n");

    printf("Port %u initialized and started\n", port_id);

    struct ping_pong_buffer ping = {.count = 0};
    struct ping_pong_buffer pong = {.count = 0};
    struct ping_pong_buffer *active_buffer = &ping;
    int is_ping_active = 1;

    char *d_buffer;
    size_t buffer_size = PING_PONG_SIZE * RTE_MBUF_DEFAULT_BUF_SIZE;
    hipMalloc((void**)&d_buffer, buffer_size);

    struct rte_mbuf *bufs[BURST_SIZE];
    uint16_t nb_rx;
    int total_drops = 0;

    while (1) {
        nb_rx = rte_eth_rx_burst(port_id, 0, bufs, BURST_SIZE);

        if (nb_rx > 0) {
            for (int i = 0; i < nb_rx; i++) {
                if (bufs[i] == NULL) {
                    printf("Mbuf allocation failed. Packet dropped.\n");
                    total_drops++;
                } else {
                    struct rte_mbuf *copy = deep_copy_mbuf(bufs[i], mbuf_pool);
                    if (copy == NULL) {
                        total_drops++;
                        continue;
                    }
                    active_buffer->pkts[active_buffer->count] = copy;
                    active_buffer->count++;
                    if (active_buffer->count >= PING_PONG_SIZE) {
                        host_to_device(active_buffer, d_buffer, buffer_size);
                        is_ping_active = !is_ping_active;
                        active_buffer = is_ping_active ? &ping : &pong;
                    }
                }
                rte_pktmbuf_free(bufs[i]);
            }

            rte_eth_stats_get(port_id, &eth_stats);
            if ((eth_stats.ipackets + eth_stats.imissed) >= TOTAL_PACKETS) break;
        }

        rte_delay_ms(sleep_ms);
    }

    printf("Packets received: %lu, Packets dropped at NIC: %lu, Total drops at application layer: %d\n",
           eth_stats.ipackets, eth_stats.imissed, total_drops);

    rte_eth_dev_stop(port_id);
    rte_eth_dev_close(port_id);
    hipFree(d_buffer);
    printf("Port %u closed\n", port_id);

    return 0;
}
